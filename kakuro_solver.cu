#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <fstream>
#include <sstream>
#include <vector>
#include <array>
#include <stack>
#include <bits/stdc++.h>
#include <math.h>

using namespace std;

enum direction
{
    d_down,
    d_right,
    none
};

#define COORD std::pair<int, int>

// #define DEBUG

int iter = 0;

//////////////////////////////////////////////
// Auxiliary functions for preparing problem //
//////////////////////////////////////////////

void display_arr(int *arr, int n)
{

    cout << "arr: ";

    for (int i = 0; i < n; i++)
    {
        cout << arr[i] << " ";
    }

    cout << endl;
}

void print_coords(COORD start, COORD end)
{

    cout << "Start:" << start.first << "," << start.second << endl;
    cout << "End:" << end.first << "," << end.second << endl;
}

int find_length(COORD start, COORD end, direction dir)
{

    if (dir == d_down)
        return end.first - start.first;
    if (dir == d_right)
        return end.second - start.second;

    return -1;
}

void convert_sol(int **mat, int **&sol_mat, int m, int n)
{

    sol_mat = new int *[m]; // Rows
    for (int i = 0; i < m; i++)
    {
        sol_mat[i] = new int[n]; // Cols
    }

    for (int i = 0; i < m; i++)
    {
        for (int j = 0; j < m; j++)
        {
            if (mat[i][j] == -2)
                sol_mat[i][j] = -2; // Empty value cell
            else
                sol_mat[i][j] = -1; // Hint or empty cell
        }
    }
}

void print_one_matrix(int **matrix, int m, int n)
{
    std::cout << "Matrix: " << std::endl;
    for (int i = 0; i < m; i++)
    { // rows
        for (int j = 0; j < n; j++)
        { // cols
            std::cout << matrix[i][j] << "\t";
        }
        std::cout << "\n";
    }
}

/// Auxiliary functions

struct sum
{
    COORD start;
    COORD end;

    int hint;
    int dir;
    int length;
    int *arr;

    void print_sum()
    {
        cout << "############################" << endl;
        cout << "Creating sum with: " << endl;
        print_coords(start, end);
        cout << "Hint: " << hint << endl;
        cout << "Direction: " << dir << endl;
        cout << "Length: " << length << endl;
        cout << "############################" << endl;
    }

    sum(COORD _start, COORD _end, int _hint, direction _dir) : start(_start), end(_end), hint(_hint), dir(_dir)
    {
        length = find_length(_start, _end, _dir);
        arr = new int[length];
#ifdef DEBUG
        cout << "############################" << endl;
        cout << "Creating sum with: " << endl;
        print_coords(start, end);
        cout << "Hint: " << hint << endl;
        cout << "Direction: " << dir << endl;
        cout << "Length: " << length << endl;
        cout << "############################" << endl;
#endif
    }

    //~sum(){
    // delete arr;
    //}
};

COORD find_end(int **matrix, int m, int n, int i, int j, direction dir)
{ // 0 down 1 right

    if (dir == d_right)
    {
        for (int jj = j + 1; jj < n; jj++)
        {
            if (matrix[i][jj] != -2 || jj == n - 1)
            {
                if (matrix[i][jj] == -2 && jj == n - 1)
                    jj++;
                COORD END = COORD(i, jj);
                return END;
            }
        }
    }

    if (dir == d_down)
    {
        for (int ii = i + 1; ii < m; ii++)
        {
            if (matrix[ii][j] != -2 || ii == m - 1)
            {
                if (matrix[ii][j] == -2 && ii == m - 1)
                    ii++;
                COORD END = COORD(ii, j);
                return END;
            }
        }
    }

    return COORD();
}

vector<sum> get_sums(int **matrix, int m, int n)
{

    vector<sum> sums;

    for (int i = 0; i < m; i++)
    {
        for (int j = 0; j < n; j++)
        {
            int val = matrix[i][j];
            if (val != -1 && val != -2)
            {
                int hint = val;
                hint = hint / 10;

                if ((hint % 100) == 0)
                {
                    hint = (int)(hint / 100);
                    COORD START = COORD(i, j + 1);
                    COORD END = find_end(matrix, m, n, i, j, d_right);
                    sum _sum = sum(START, END, hint, d_right);
                    sums.push_back(_sum);
                }

                else
                {
                    int div = (int)(hint / 100);
                    int rem = (int)(hint % 100);

                    if (div == 0 && rem != 0)
                    {
                        COORD START = COORD(i + 1, j);
                        COORD END = find_end(matrix, m, n, i, j, d_down);
                        sum _sum = sum(START, END, rem, d_down);
                        sums.push_back(_sum);
                    }

                    if (div != 0 && rem != 0)
                    {
                        COORD START1 = COORD(i + 1, j);
                        COORD START2 = COORD(i, j + 1);
                        COORD END1 = find_end(matrix, m, n, i, j, d_down);
                        COORD END2 = find_end(matrix, m, n, i, j, d_right);
                        sum _sum1 = sum(START1, END1, rem, d_down);
                        sum _sum2 = sum(START2, END2, div, d_right);
                        sums.push_back(_sum1);
                        sums.push_back(_sum2);
                    }
                }
            }
        }
    }
    return sums;
}

void read_matrix(int **&matrix, std::ifstream &afile, int m, int n)
{

    matrix = new int *[m]; // rows

    for (int i = 0; i < m; i++)
    {
        matrix[i] = new int[n]; // cols
    }

    int val;
    for (int i = 0; i < m; i++)
    {
        for (int j = 0; j < n; j++)
        {
            afile >> val;
            matrix[i][j] = val;
        }
    }
}

void sol_to_file(int **mat, int **sol_mat, int m, int n)
{

    string fname = "visualize.kakuro";
    ofstream to_write(fname);

    to_write << m << " " << n << "\n";

    for (int i = 0; i < m; i++)
    {
        for (int j = 0; j < n; j++)
        {
            if (mat[i][j] != -2)
                to_write << mat[i][j] << " ";
            else
                to_write << sol_mat[i][j] << " ";
        }
        to_write << "\n";
    }

    to_write.close();
}

//////////////////////////////////////////////
// Auxiliary functions for preparing problem //
//////////////////////////////////////////////

///////////////////////////////////////////////////
// Auxiliary functions for preparing CUDA setting //
///////////////////////////////////////////////////

void flatten_sums(vector<sum> sums, int *h_sum_starts_x, int *h_sum_starts_y, int *h_sum_ends_x, int *h_sum_ends_y, int *h_sum_hints, int *h_sum_lengths, int *h_sum_dirs, int no_sums)
{

    for (int i = 0; i < no_sums; i++)
    {

        h_sum_starts_x[i] = sums[i].start.first;
        h_sum_starts_y[i] = sums[i].start.second;

        h_sum_ends_x[i] = sums[i].end.first;
        h_sum_ends_y[i] = sums[i].end.second;

        h_sum_hints[i] = sums[i].hint;
        h_sum_lengths[i] = sums[i].length;

        h_sum_dirs[i] = sums[i].dir;
    }
}

void print_flattened(int *h_sum_starts_x, int *h_sum_starts_y, int *h_sum_ends_x, int *h_sum_ends_y, int *h_sum_hints, int *h_sum_lengths, int *h_sum_dirs, int no_sums)
{

    cout << "###h_sum_starts_x: " << endl;
    for (int i = 0; i < no_sums; i++)
    {
        cout << h_sum_starts_x[i] << " ";
    }
    cout << endl;

    cout << "###h_sum_starts_y: " << endl;
    for (int i = 0; i < no_sums; i++)
    {
        cout << h_sum_starts_y[i] << " ";
    }
    cout << endl;

    cout << "###h_sum_ends_x: " << endl;
    for (int i = 0; i < no_sums; i++)
    {
        cout << h_sum_ends_x[i] << " ";
    }
    cout << endl;

    cout << "###h_sum_ends_y: " << endl;
    for (int i = 0; i < no_sums; i++)
    {
        cout << h_sum_ends_y[i] << " ";
    }
    cout << endl;

    cout << "###h_sum_hints: " << endl;
    for (int i = 0; i < no_sums; i++)
    {
        cout << h_sum_hints[i] << " ";
    }
    cout << endl;

    cout << "###h_sum_lengths: " << endl;
    for (int i = 0; i < no_sums; i++)
    {
        cout << h_sum_lengths[i] << " ";
    }
    cout << endl;

    cout << "###h_sum_dirs: " << endl;
    for (int i = 0; i < no_sums; i++)
    {
        cout << h_sum_dirs[i] << " ";
    }
    cout << endl;
}

void flatten_sol_mat(int **sol_mat, int *h_sol_mat, int m, int n)
{
    for (int i = 0; i < m; i++)
    {
        for (int j = 0; j < n; j++)
        {
            h_sol_mat[i * n + j] = sol_mat[i][j];
        }
    }
}

void print_flattened_matrix(int *h_sol_mat, int m, int n)
{

    cout << "###Flattened matrix: " << endl;
    for (int i = 0; i < m; i++)
    {
        for (int j = 0; j < n; j++)
        {
            cout << h_sol_mat[i * n + j] << " ";
        }
        cout << endl;
    }
    cout << endl;
}

///////////////////////////////////////////////////
// Auxiliary functions for preparing CUDA setting //
///////////////////////////////////////////////////

///////////////////
// CUDA FUNCTIONS //
///////////////////

// For debugging.
__device__ void print_device_matrix(int **mat, int m, int n)
{
    for (int i = 0; i < m; i++)
    {
        for (int j = 0; j < n; j++)
        {
            printf(" %d", mat[i][j]);
        }
        printf("\n");
    }
}

// For debugging.
__device__ void print_device_matrix(int *mat, int size)
{
    for (int i = 0; i < size; i++)
    {
        printf(" %d", mat[i]);
    }
}

__device__ bool checkSumStatus(int remaining_sum, int remaining_cells)
{
    int current_max_num = 9;
    int current_min_num = 1;
    int max_num = 0;
    int min_num = 0;

    for (int i = 0; i < remaining_cells; i++)
    {
        max_num += current_max_num;
        min_num += current_min_num;
        current_max_num--;
        current_min_num++;
    }

    // remaining_sum > maximum value that can fit into remaining_cells.
    // We need to put bigger values to cells: anything containing smaller nums will be wrong
    if (remaining_sum > max_num)
        return false;

    // remaining_sum < minimum value that can fit into remaining_cells.
    // We need to put smaller values to cells: anything containing bigger nums will be wrong
    if (remaining_sum < min_num)
        return false;

    return true;
}

// Checks the solution matrix whether it is valid or has potential to be valid for a given sum object.
// It also checks for duplicates.
__device__ bool checkSum(int *&d_sum_starts_x, int *&d_sum_starts_y, int *&d_sum_ends_x, int *&d_sum_ends_y,
                         int *&d_sum_hints, int *&d_sum_dirs, int *&board, int m,
                         int n, int d_sum_idx, int k)
{
    // If there is no sum, don't check.
    if (d_sum_idx == -1)
    {
        return false;
    }

    int hint = d_sum_hints[d_sum_idx];

    int row_idx = d_sum_starts_x[d_sum_idx];
    int col_idx = d_sum_starts_y[d_sum_idx];

    // Check for a row sum
    if (d_sum_dirs[d_sum_idx] == 1)
    {
        int end_idx = d_sum_ends_y[d_sum_idx];

        // Continue iteration until there is a currently empty cell or end of the sum region.
        while (col_idx < end_idx && board[(row_idx * m) + col_idx] > 0)
        {
            // Substract the remaining sum by the value inside the sum region.
            hint -= board[(row_idx * m) + col_idx];
            bool status = checkSumStatus(hint, end_idx - col_idx - 1);
            // If sum status is not valid, return false.
            if (!status)
            {
                return false;
            }

            // Check for duplicates.
            if ((row_idx * m) + col_idx != k && board[k] == board[(row_idx * m) + col_idx])
            {
                return false;
            }

            col_idx++;
        }
    }

    // Check for a column sum
    else
    {
        int end_idx = d_sum_ends_x[d_sum_idx];

        // Continue iteration until there is a currently empty cell or end of the sum region.

        while (row_idx < end_idx && board[(row_idx * m) + col_idx] > 0)
        {
            // Substract the remaining sum by the value inside the sum region.
            hint -= board[(row_idx * m) + col_idx];
            bool status = checkSumStatus(hint, end_idx - row_idx - 1);

            // If sum status is not valid, return false.
            if (!status)
            {
                return false;
            }

            // Check for duplicates.
            if ((row_idx * m) + col_idx != k && board[k] == board[(row_idx * m) + col_idx])
            {

                return false;
            }

            row_idx++;
        }
    }
    return true;
}

// 2D array to map board cells to the flattened sum array indexes they are included in.
__device__ int **setCell2SumIdx(int *&d_sum_starts_x, int *&d_sum_starts_y, int *&d_sum_ends_x, int *&d_sum_ends_y,
                                int *&d_sum_dirs, int m, int n, int d_sum_count)
{
    int **cell_2_sum_idx = new int *[m * n];

    for (int i = 0; i < m * n; i++)
    {
        cell_2_sum_idx[i] = new int[2];
        for (int j = 0; j < 2; j++)
        {
            cell_2_sum_idx[i][j] = -1;
        }
    }

    for (int i = 0; i < d_sum_count; i++)
    {
        int start_row = d_sum_starts_x[i];
        int start_col = d_sum_starts_y[i];
        int end_row = d_sum_ends_x[i];
        int end_col = d_sum_ends_y[i];

        int start_k = start_row * m + start_col;
        int end_k = end_row * m + end_col;

        if (d_sum_dirs[i] == direction::d_right)
        {
            for (int j = start_k; j < end_k; j++)
            {
                if (cell_2_sum_idx[j][0] == -1) // If first sum
                {
                    cell_2_sum_idx[j][0] = i;
                }
                else
                {
                    cell_2_sum_idx[j][1] = i;
                }
            }
        }
        else
        {
            for (int j = start_k; j < end_k; j += m)
            {
                if (cell_2_sum_idx[j][0] == -1) // If first sum
                {
                    cell_2_sum_idx[j][0] = i;
                }
                else
                {
                    cell_2_sum_idx[j][1] = i;
                }
            }
        }
    }

    return cell_2_sum_idx;
}

// Generate deep copy of a matrix.
__device__ int **copyMatrix(int **mat, int m, int n)
{
    int **copy = new int *[m];
    for (int i = 0; i < m; i++)
    {
        copy[i] = new int[n];
        for (int j = 0; j < n; j++)
        {
            copy[i][j] = mat[i][j];
        }
    }

    return copy;
}

// Generate deep copy of a flatted matrix.
__device__ int *copyMatrixFlattened(int *mat, int size)
{
    int *copy = new int[size];
    for (int i = 0; i < size; i++)
    {
        copy[i] = mat[i];
    }
    return copy;
}

// Task generator kernel.
// Generates possibly valid boards from current tasks i.e. current boards.
// Each block is responsible for generating and checking 9 boards from previous boards. So, block_size = num_tasks.
// Each thread is responsible for a single board by inserting threadIdx.x + 1 to the next cell in the board.
// Store them in 2d array: new_tasks.
__global__ void kakuro_solver(int *d_sum_starts_x, int *d_sum_starts_y, int *d_sum_ends_x, int *d_sum_ends_y,
                              int *d_sum_hints, int *d_sum_dirs, int *d_sol_mat, int **tasks,
                              int m, int n, int k, int **d_cell2sum_idx, int **new_tasks, int dim)
{
    // Copy a board regarding to blockIdx.
    int *board = copyMatrixFlattened(tasks[blockIdx.x], m * n);

    // Insert a new number to board regarding to threadIdx.
    int num = threadIdx.x + 1;
    board[k] = num;

    // Get sum indexes from the map.
    int sum_idx_1 = d_cell2sum_idx[k][0];
    int sum_idx_2 = d_cell2sum_idx[k][1];
    bool status;

    // Check for the first sum.
    status = checkSum(d_sum_starts_x, d_sum_starts_y, d_sum_ends_x, d_sum_ends_y, d_sum_hints, d_sum_dirs, board, m, n, sum_idx_1, k);

    if (!status)
    {
        new_tasks[blockDim.x * blockIdx.x + threadIdx.x] = nullptr;
        delete[] board;
        return;
    }

    // Check for the first sum.
    status = checkSum(d_sum_starts_x, d_sum_starts_y, d_sum_ends_x, d_sum_ends_y, d_sum_hints, d_sum_dirs, board, m, n, sum_idx_2, k);
    if (!status)
    {
        new_tasks[blockDim.x * blockIdx.x + threadIdx.x] = nullptr;
        delete[] board;
        return;
    }

    // If no errors are present, generate a new task from the current state of the board.
    new_tasks[blockDim.x * blockIdx.x + threadIdx.x] = board;
}

// Kakuro solver controller kernel.
// This kernel runs with a single block and single thread.
// Since kakuro_solver is called multiple times for each step, it needs a controller inside the GPU.
// So that memory transfer does not cost as much as controlling it from the CPU.
__global__ void kakuro_kernel(int *d_sum_starts_x, int *d_sum_starts_y, int *d_sum_ends_x, int *d_sum_ends_y,
                              int *d_sum_hints, int *d_sum_dirs, int *d_sol_mat, int m, int n, int no_sums)
{
    int **cell_2_sum_idx = setCell2SumIdx(d_sum_starts_x, d_sum_starts_y, d_sum_ends_x, d_sum_ends_y,
                                          d_sum_dirs, m, n, no_sums);

    // Start with a single board.
    int num_tasks = 1;
    int **tasks = new int *[num_tasks];

    tasks[0] = copyMatrixFlattened(d_sol_mat, m * n);

    for (int k = 0; k < m * n; k++)
    {
        if (tasks[0][k] != -2)
            continue;

        // Maximum size of tasks that will be generated is 9 * num_tasks, allocate memory for that.
        int num_new_tasks = 9 * num_tasks;
        int **new_tasks = new int *[num_new_tasks];

#ifdef DEBUG
        printf("NUM BLOCKS STEP %d: %d\n", k, num_tasks);
#endif

        // Run kakuro_solver for the current tasks / boards.
        kakuro_solver<<<num_tasks, 9>>>(d_sum_starts_x, d_sum_starts_y, d_sum_ends_x, d_sum_ends_y,
                                        d_sum_hints, d_sum_dirs, d_sol_mat, tasks, m, n, k, cell_2_sum_idx, new_tasks, num_tasks);

        // Wait for kakuro_solver to finish.
        hipDeviceSynchronize();

        // Organize and reset tasks and new tasks:

        // Delete previous tasks:
        for (int i = 0; i < num_tasks; i++)
        {
            delete[] tasks[i];
        }
        delete[] tasks;

        // Count new number of tasks.
        num_tasks = 0;
        for (int i = 0; i < num_new_tasks; i++)
        {
            if (new_tasks[i])
                num_tasks++;
        }

        // Allocate memory for new tasks.
        tasks = new int *[num_tasks];
        int task_idx = 0;

        for (int i = 0; i < num_new_tasks; i++)
        {
            if (new_tasks[i])
            {
                tasks[task_idx] = new_tasks[i];
                task_idx++;
            }
        }
        delete[] new_tasks;
    }

    for (int i = 0; i < m * n; i++)
    {
        d_sol_mat[i] = tasks[0][i];
    }
#ifdef DEBUG
    printf("SOL HERE\n");
    print_device_matrix(d_sol_mat, m * n);
    printf("\n\n");
#endif
}

///////////////////
// CUDA FUNCTIONS //
///////////////////

// Write solution to file.
void sol_mat_flattened_to_file(int **mat, int *d_sol_mat, int m, int n, string fname)
{
    ofstream to_write(fname);
    to_write << m << " " << n << "\n";

    int *h_sol_mat_f = new int[m * n];
    hipMemcpy(h_sol_mat_f, d_sol_mat, m * n * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < m; i++)
    {
        for (int j = 0; j < n; j++)
        {
            if (mat[i][j] != -2)
                to_write << mat[i][j] << " ";
            else
                to_write << h_sol_mat_f[i * m + j] << " ";
        }
        to_write << "\n";
    }

    to_write.close();
}

int main(int argc, char **argv)
{

    std::string filename(argv[1]);
    std::ifstream file;
    file.open(filename.c_str());

    int m, n;
    file >> m;
    file >> n;

    int **mat;
    read_matrix(mat, file, m, n);
    print_one_matrix(mat, m, n);

    int **sol_mat;
    convert_sol(mat, sol_mat, m, n);
    // print_one_matrix(sol_mat, m, n);

    vector<sum> sums = get_sums(mat, m, n);

    // CUDA
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    printf("==prop== Running on device: %d -- %s \n", 0, prop.name);
    printf("==prop== #of SM -- %d \n", prop.multiProcessorCount);
    printf("==prop== Max Threads Per Block: -- %d \n", prop.maxThreadsPerBlock);

    int grid_dim = 1;
    int block_dim = 1;

    int no_sums = sums.size();

    // Flattening sums and matrix
    int *h_sum_starts_x = new int[no_sums];
    int *h_sum_starts_y = new int[no_sums];
    int *h_sum_ends_x = new int[no_sums];
    int *h_sum_ends_y = new int[no_sums];
    int *h_sum_hints = new int[no_sums];
    int *h_sum_lengths = new int[no_sums];
    int *h_sum_dirs = new int[no_sums];

    flatten_sums(sums, h_sum_starts_x, h_sum_starts_y, h_sum_ends_x, h_sum_ends_y, h_sum_hints, h_sum_lengths, h_sum_dirs, no_sums);

    print_flattened(h_sum_starts_x, h_sum_starts_y, h_sum_ends_x, h_sum_ends_y, h_sum_hints, h_sum_lengths, h_sum_dirs, no_sums);

    int *h_sol_mat;
    h_sol_mat = new int[m * n];
    flatten_sol_mat(sol_mat, h_sol_mat, m, n);

    print_flattened_matrix(h_sol_mat, m, n);

    // Declare device pointers and copy data into device
    int *d_sum_starts_x, *d_sum_starts_y, *d_sum_ends_x, *d_sum_ends_y, *d_sum_hints, *d_sum_lengths, *d_sum_dirs, *d_sol_mat, *d_t_mats;

    hipMalloc(&d_sum_starts_x, no_sums * sizeof(int));
    hipMalloc(&d_sum_starts_y, no_sums * sizeof(int));
    hipMalloc(&d_sum_ends_x, no_sums * sizeof(int));
    hipMalloc(&d_sum_ends_y, no_sums * sizeof(int));
    hipMalloc(&d_sum_hints, no_sums * sizeof(int));
    hipMalloc(&d_sum_lengths, no_sums * sizeof(int));
    hipMalloc(&d_sum_dirs, no_sums * sizeof(int));
    hipMalloc(&d_sol_mat, (m * n) * sizeof(int));
    hipMalloc(&d_t_mats, (m * n * grid_dim * block_dim) * sizeof(int)); // Allocating invidual matrix for each GPU thread
    // You may use this array if you will implement a thread-wise solution

    hipMemcpy(d_sum_starts_x, h_sum_starts_x, no_sums * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_sum_starts_y, h_sum_starts_y, no_sums * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_sum_ends_x, h_sum_ends_x, no_sums * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_sum_ends_y, h_sum_ends_y, no_sums * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_sum_hints, h_sum_hints, no_sums * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_sum_lengths, h_sum_lengths, no_sums * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_sum_dirs, h_sum_dirs, no_sums * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_sol_mat, h_sol_mat, (m * n) * sizeof(int), hipMemcpyHostToDevice);

    // ALLOCATE 8GB
    size_t rsize = 1024ULL * 1024ULL * 1024ULL * 8ULL;
    hipDeviceSetLimit(hipLimitMallocHeapSize, rsize);

    kakuro_kernel<<<grid_dim, block_dim>>>(d_sum_starts_x, d_sum_starts_y, d_sum_ends_x, d_sum_ends_y, d_sum_hints,
                                           d_sum_dirs, d_sol_mat, m, n, no_sums);
    hipDeviceSynchronize();
    // CUDA

    int *h_sol_mat_f = new int[m * n];
    hipMemcpy(h_sol_mat_f, d_sol_mat, m * n * sizeof(int), hipMemcpyDeviceToHost);
    print_flattened_matrix(h_sol_mat_f, m, n);

    string fname = argv[1];
    fname = fname.substr(0, fname.length() - 7) + "_solution.kakuro";

    sol_mat_flattened_to_file(mat, d_sol_mat, m, n, fname);
    // Similiar to sol_mat, use hints from mat and values from d_sol_mat

    for (int i = 0; i < n; i++)
    {
        delete mat[i];
        delete sol_mat[i];
    }

    delete mat;
    delete sol_mat;

    delete h_sum_starts_x;
    delete h_sum_starts_y;
    delete h_sum_ends_x;
    delete h_sum_ends_y;
    delete h_sum_hints;
    delete h_sum_lengths;
    delete h_sum_dirs;
    delete h_sol_mat;

    hipFree(d_t_mats);
    hipFree(d_sum_starts_x);
    hipFree(d_sum_starts_y);
    hipFree(d_sum_ends_x);
    hipFree(d_sum_ends_y);
    hipFree(d_sum_hints);
    hipFree(d_sum_lengths);
    hipFree(d_sum_dirs);
    hipFree(d_sol_mat);

    return 0;
}
